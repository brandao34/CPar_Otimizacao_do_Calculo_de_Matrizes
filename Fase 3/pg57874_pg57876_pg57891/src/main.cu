#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include "EventManager.h"
#include "fluid_solver.h"

#define SIZE 168

#define IX(i, j, k) ((i) + (SIZE + 2) * (j) + (SIZE + 2) * (SIZE + 2) * (k))

// Globals for the grid size
static int M = SIZE;
static int N = SIZE;
static int O = SIZE;
static float dt = 0.1f;       // Time delta
static float diff = 0.0001f;  // Diffusion constant
static float visc = 0.0001f;  // Viscosity constant

// Fluid simulation arrays
static float *u, *v, *w, *u_prev, *v_prev, *w_prev;
static float *dens, *dens_prev;

// Function to allocate simulation data
int allocate_data() {
    int size = (M + 2) * (N + 2) * (O + 2);
    hipMalloc((void**)&u, size * sizeof(float));
    hipMalloc((void**)&v, size * sizeof(float));
    hipMalloc((void**)&w, size * sizeof(float));
    hipMalloc((void**)&u_prev, size * sizeof(float));
    hipMalloc((void**)&v_prev, size * sizeof(float));
    hipMalloc((void**)&w_prev, size * sizeof(float));
    hipMalloc((void**)&dens, size * sizeof(float));
    hipMalloc((void**)&dens_prev, size * sizeof(float));

    if (!u || !v || !w || !u_prev || !v_prev || !w_prev || !dens || !dens_prev) {
        std::cerr << "Cannot allocate memory" << std::endl;
        return 0;
    }
    return 1;
}

// Function to clear the data (set all to zero)
void clear_data() {
    int size = (M + 2) * (N + 2) * (O + 2);
    hipMemset(u, 0, size * sizeof(float));
    hipMemset(v, 0, size * sizeof(float));
    hipMemset(w, 0, size * sizeof(float));
    hipMemset(u_prev, 0, size * sizeof(float));
    hipMemset(v_prev, 0, size * sizeof(float));
    hipMemset(w_prev, 0, size * sizeof(float));
    hipMemset(dens, 0, size * sizeof(float));
    hipMemset(dens_prev, 0, size * sizeof(float));
}


// Free allocated memory
void free_data() {
    hipFree(u);
    hipFree(v);
    hipFree(w);
    hipFree(u_prev);
    hipFree(v_prev);
    hipFree(w_prev);
    hipFree(dens);
    hipFree(dens_prev);
}

// Apply events (source or force) for the current timestep
__global__ void apply_events_kernel(float *dens, float *u, float *v, float *w, Event *events, int num_events) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_events) {
        Event event = events[idx];
        int i = SIZE / 2, j = SIZE / 2, k = SIZE / 2;
        if (event.type == ADD_SOURCE) {
            dens[IX(i, j, k)] = event.density;
        } else if (event.type == APPLY_FORCE) {
            u[IX(i, j, k)] = event.force.x;
            v[IX(i, j, k)] = event.force.y;
            w[IX(i, j, k)] = event.force.z;
        }
    }
}

void apply_events(const std::vector<Event> &events) {
    Event *d_events;
    hipMalloc(&d_events, events.size() * sizeof(Event));
    hipMemcpy(d_events, events.data(), events.size() * sizeof(Event), hipMemcpyHostToDevice);

    int blockSize = 128;
    int numBlocks = (events.size() + blockSize - 1) / blockSize;
    apply_events_kernel<<<numBlocks, blockSize>>>(dens, u, v, w, d_events, events.size());

    hipFree(d_events);
}

// Function to sum the total density
float sum_density() {
    int size = (M + 2) * (N + 2) * (O + 2);

    std::vector<float> temp(size);

    hipMemcpy(temp.data(), dens, size * sizeof(float), hipMemcpyDeviceToHost);

    float total = 0;
    for (int i = 0; i < size; i++) {
        total += temp[i];
    }
    return total;
}

// Simulation loop
void simulate(EventManager &eventManager, int timesteps) {
    for (int t = 0; t < timesteps; t++) {
        // Get the events for the current timestep
        std::vector<Event> events = eventManager.get_events_at_timestamp(t);

        // Apply events to the simulation
        apply_events(events);

        // Perform the simulation steps
        vel_step(M, N, O, u, v, w, u_prev, v_prev, w_prev, visc, dt);
        dens_step(M, N, O, dens, dens_prev, u, v, w, diff, dt);
    }
}

int main() {
    // Initialize EventManager
    EventManager eventManager;
    eventManager.read_events("events.txt");

    // Get the total number of timesteps from the event file
    int timesteps = eventManager.get_total_timesteps();

    // Allocate and clear data
    if (!allocate_data())
        return -1;
    clear_data();

    // Run simulation with events
    simulate(eventManager, timesteps);

    // Print total density at the end of simulation
    float total_density = sum_density();
    std::cout << "Total density after " << timesteps
              << " timesteps: " << total_density << std::endl;

    // Free memory
    free_data();

    return 0;
}